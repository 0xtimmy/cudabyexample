#include "hip/hip_runtime.h"
#include<stdio.h>

#define N (1024*1024)
#define FULL_DATA_SIZE (N*20)

__global__ void kernel(int *a, int *b, int *c);

int main ( void ) {
    hipDeviceProp_t prop;
    hipGetDeviceProperties ( &prop, 0 );
    if(!prop.deviceOverlap) {
        printf("Device will not handle system overlaps :(\n");
        return 0;
    } else {
        printf("Device will handle system overlaps :)\n");
    }

    hipEvent_t start, stop;
    float elapsedTime;
    hipEventCreate( &start );
    hipEventCreate( &stop );
    hipEventRecord(start, 0);

    hipStream_t stream0, stream1;

    hipStreamCreate( &stream0 );
    hipStreamCreate( &stream1 );

    int *host_a, *host_b, *host_c;
    int *dev_a0, *dev_b0, *dev_c0;
    int *dev_a1, *dev_b1, *dev_c1;

    hipMalloc((void**)&dev_a0, N*sizeof(int));
    hipMalloc((void**)&dev_b0, N*sizeof(int));
    hipMalloc((void**)&dev_c0, N*sizeof(int));
    hipMalloc((void**)&dev_a1, N*sizeof(int));
    hipMalloc((void**)&dev_b1, N*sizeof(int));
    hipMalloc((void**)&dev_c1, N*sizeof(int));

    hipHostAlloc((void**)&host_a, FULL_DATA_SIZE * sizeof(int), hipHostMallocDefault);
    hipHostAlloc((void**)&host_b, FULL_DATA_SIZE * sizeof(int), hipHostMallocDefault);
    hipHostAlloc((void**)&host_c, FULL_DATA_SIZE * sizeof(int), hipHostMallocDefault);

    for (int i = 0; i < FULL_DATA_SIZE; i++) {
        host_a[i] = rand();
        host_b[i] = rand();
    }

    for(int i = 0; i < FULL_DATA_SIZE; i += N*2) {
        hipMemcpyAsync(dev_a0, host_a+i, N * sizeof(int), hipMemcpyHostToDevice, stream0);
        hipMemcpyAsync(dev_a1, host_a+i+N, N * sizeof(int), hipMemcpyHostToDevice, stream1);
        hipMemcpyAsync(dev_b0, host_b+i, N * sizeof(int), hipMemcpyHostToDevice, stream0);
        hipMemcpyAsync(dev_b1, host_b+i+N, N * sizeof(int), hipMemcpyHostToDevice, stream1);

        kernel<<<N/256,256,0,stream0>>>( dev_a0, dev_b0, dev_c0);
        kernel<<<N/256,256,0,stream0>>>( dev_a1, dev_b1, dev_c1);

        hipMemcpyAsync(host_c+i, dev_c0, N * sizeof(int), hipMemcpyDeviceToHost, stream0);
        hipMemcpyAsync(host_c+i+N, dev_c1, N * sizeof(int), hipMemcpyDeviceToHost, stream1);
    }

    hipStreamSynchronize(stream0);
    hipStreamSynchronize(stream1);

    hipEventRecord(stop, 0);
    hipEventSynchronize(stop);
    hipEventElapsedTime(&elapsedTime, start, stop);
    
    printf("Time taken: %3.1fms\n", elapsedTime);

    hipHostFree(host_a);
    hipHostFree(host_b);
    hipHostFree(host_c);
    hipFree(dev_a0);
    hipFree(dev_b0);
    hipFree(dev_c0);
    hipFree(dev_a1);
    hipFree(dev_b1);
    hipFree(dev_c1);

    hipStreamDestroy(stream0);
    hipStreamDestroy(stream1);
    return 0;
}


__global__ void kernel(int *a, int *b, int *c) {
    int idx = threadIdx.x + blockIdx.x * blockDim.x;
    if (idx < N) {
        int idx1 = (idx + 1) % 256;
        int idx2 = (idx + 2) % 256;

        float as = (a[idx] + a[idx1] + a[idx2]) / 3.0f;
        float bs = (b[idx] + b[idx1] + b[idx2]) / 3.0f;
        c[idx] = (as + bs) / 2;
    }
}