
#include <hip/hip_runtime.h>
#include<stdio.h>

#define imin(a,b) (a<b?a:b)
#define sum_squares(x) (x*(x+1)*(2*x+1)/6)

const int N = 33 * 1024;
const int threadsPerBlock = 256;
const int blocksPerGrid = imin (32, (N+threadsPerBlock-1)/threadsPerBlock);


__global__ void dot( float *a, float *b, float *c) {
    __shared__ float cache[threadsPerBlock];
    int tid = threadIdx.x + blockIdx.x * blockDim.x;
    int cacheIndex = threadIdx.x;
    float temp = 0;
    while (tid < N) {
        temp += a[tid] * b[tid];
        tid += blockDim.x * gridDim.x;
    }

    cache[cacheIndex] = temp;

    __syncthreads();

    int i = blockDim.x/2;
    while (i != 0) {
        if (cacheIndex < i) {
            cache[cacheIndex] += cache[cacheIndex + i];
        }
        __syncthreads();
        i /= 2;
    }

    c[blockIdx.x] = cache[0];
}

int main ( void ) {
    float *a, *b, c, *partial_c;
    float *dev_a, *dev_b, *dev_partial_c;

    a = new float[N];
    b = new float[N];
    partial_c = new float[blocksPerGrid];

    hipMalloc((void**) &dev_a, N*sizeof(float));
    hipMalloc((void**) &dev_b, N*sizeof(float));
    hipMalloc((void**) &dev_partial_c, blocksPerGrid*sizeof(float));

    for (int i = 0; i < N; i++) {
        a[i] = i;
        b[i] = i*2;
    }

    hipMemcpy(dev_a, a, N*sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(dev_b, b, N*sizeof(float), hipMemcpyHostToDevice);

    dot<<<blocksPerGrid,threadsPerBlock>>>(dev_a, dev_b, dev_partial_c);

    hipMemcpy(partial_c, dev_partial_c, blocksPerGrid*sizeof(float), hipMemcpyDeviceToHost);

    c = 0;
    for(int i = 0; i < blocksPerGrid; i++) {
        c += partial_c[i];
    }

    printf("Does GPU value%.6g = %.6g?\n", c, 2 * sum_squares((float) (N-1)));

    hipFree(dev_a);
    hipFree(dev_b);
    hipFree(dev_partial_c);

    delete [] a;
    delete [] b;
    delete [] partial_c;
}