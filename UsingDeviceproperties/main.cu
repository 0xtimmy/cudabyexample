
#include <hip/hip_runtime.h>
#include<stdio.h>

int main( void ) {
    hipDeviceProp_t prop;
    int dev;

    hipGetDevice( &dev );
    printf("Id of current device: %d\n", dev);

    memset ( &prop, 0, sizeof( hipDeviceProp_t ));
    prop.major = 8;
    prop.minor = 7;
    hipChooseDevice(&dev, &prop);
    printf("Id of CUDA device closest to revision %d.%d: %d\n", prop.major, prop.minor, dev);
    hipSetDevice(dev);
}