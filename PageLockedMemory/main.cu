
#include <hip/hip_runtime.h>
#include<stdio.h>
#define SIZE (10*1024*1024)

float cuda_malloc_test(int size, bool up);
float cuda_host_malloc_test(int size, bool up);

int main ( void ) {
    float elapsedTime;
    float MB = (float) 100 * SIZE * sizeof(int)/1024/1024;

    elapsedTime = cuda_malloc_test( SIZE, true );
    printf("Time using malloc: %3.1f\n", elapsedTime);
    printf("MB/s during copy up: %3.1f\n\n", MB/elapsedTime/1000);
    elapsedTime = cuda_malloc_test( SIZE, false );
    printf("Time using malloc: %3.1f\n", elapsedTime);
    printf("\tMB/s during copy down: %3.1f\n\n", MB/elapsedTime/1000);

    elapsedTime = cuda_host_malloc_test( SIZE, true );
    printf("Time using cudaMalloc: %3.1f\n", elapsedTime);
    printf("\tMB/s during copy up: %3.1f\n\n", MB/elapsedTime/1000);
    elapsedTime = cuda_host_malloc_test( SIZE, false );
    printf("Time using cudaMalloc: %3.1f\n", elapsedTime);
    printf("\tMB/s during copy down: %3.1f\n\n", MB/elapsedTime/1000);
    
}

float cuda_malloc_test( int size, bool up) {
    hipEvent_t start, stop;
    int *a, *dev_a;
    float elapsedTime;

    hipEventCreate( &start );
    hipEventCreate( &stop );

    hipEventRecord( start, 0 );
    a = (int *)malloc( size * sizeof( *a ));
    hipMalloc( (void**)&dev_a, size *sizeof( *dev_a ));

    for (int i = 0; i < 100; i++) {
        if(up) hipMemcpy(dev_a, a, size * sizeof( *dev_a ), hipMemcpyHostToDevice);
        else hipMemcpy(a, dev_a, size * sizeof( *dev_a ), hipMemcpyDeviceToHost);
    }
    hipEventRecord( stop, 0 );
    hipEventSynchronize( stop );
    hipEventElapsedTime( &elapsedTime, start, stop );
    free( a );
    hipFree(dev_a);
    hipEventDestroy(start);
    hipEventDestroy(stop);

    return elapsedTime;
}

float cuda_host_malloc_test( int size, bool up) {
    hipEvent_t start, stop;
    int *a, *dev_a;
    float elapsedTime;

    hipEventCreate( &start );
    hipEventCreate( &stop );

    hipEventRecord( start, 0 );
    hipHostAlloc( (void**)&a, size * sizeof( *a ), hipHostMallocDefault);
    hipMalloc( (void**)&dev_a, size *sizeof( *dev_a ));

    for (int i = 0; i < 100; i++) {
        if(up) hipMemcpy(dev_a, a, size * sizeof( *dev_a ), hipMemcpyHostToDevice);
        else hipMemcpy(a, dev_a, size * sizeof( *dev_a ), hipMemcpyDeviceToHost);
    }
    hipEventRecord( stop, 0 );
    hipEventSynchronize( stop );
    hipEventElapsedTime( &elapsedTime, start, stop );

    hipHostFree( a );
    hipFree(dev_a);
    hipEventDestroy(start);
    hipEventDestroy(stop);

    return elapsedTime;
}