#include "hip/hip_runtime.h"
#include<stdio.h>
#include<sys/time.h>
#include "../common/book.h"
#define SIZE (100*1024*1024)

__global__ void histo_kernel(unsigned char *buffer, long size, unsigned int *histo);

int main( void ) {
    unsigned char *buffer = (unsigned char*)big_random_block(SIZE, 256);

    hipEvent_t start, stop;
    hipEventCreate( &start );
    hipEventCreate( &stop );
    hipEventRecord( start, 0);

    unsigned char *dev_buffer;
    unsigned int *dev_histo;

    hipMalloc((void**)&dev_buffer, SIZE);
    hipMemcpy(dev_buffer, buffer, SIZE, hipMemcpyHostToDevice);

    hipMalloc((void**)&dev_histo, 256*sizeof(long));
    hipMemset(dev_histo, 0, 256 * sizeof(int));

    hipDeviceProp_t prop;
    hipGetDeviceProperties( &prop, 0 );
    int blocks = prop.multiProcessorCount;
    histo_kernel<<<blocks*2,256>>>(dev_buffer, SIZE, dev_histo);

    unsigned int histo[256];
    hipMemcpy(histo, dev_histo, 256 * sizeof(int), hipMemcpyDeviceToHost);

    hipEventRecord( stop, 0 );
    hipEventSynchronize( stop );
    float elapsedTime;
    hipEventElapsedTime( &elapsedTime, start, stop );
    

    timeval cpustop, cpustart;
    gettimeofday(&cpustart, NULL);
    for(int i = 0; i < SIZE; i++) {
        histo[buffer[i]]--;
    }
    gettimeofday(&cpustop, NULL);
    for (int i = 0; i < 256; i++) {
        if (histo[i] > 0) {
            printf("Failure! @ i = %d\n", i);
            break;
        }
    }

    printf("CPU took: %3.1fms\n", (float)(cpustop.tv_sec - cpustart.tv_sec) * 1000.0 + (float)(cpustop.tv_usec - cpustart.tv_usec) / 1000.0);
    printf("GPU took: %3.1fms\n", elapsedTime);

    hipEventDestroy(start);
    hipEventDestroy(stop);
    hipFree(dev_histo);
    hipFree(dev_buffer);
    free(buffer);

    return 0;
}

__global__ void histo_kernel(unsigned char *buffer, long size, unsigned int *histo) {
    
    __shared__ unsigned int temp[256];
    temp[threadIdx.x] = 0;
    __syncthreads();
    
    int i = threadIdx.x + blockIdx.x * blockDim.x;
    int offset = blockDim.x * gridDim.x;

    while(i < size) {
        atomicAdd( &temp[buffer[i]], 1);
        i += offset;
    }

    __syncthreads();

    atomicAdd( &(histo[threadIdx.x]), temp[threadIdx.x]);
}